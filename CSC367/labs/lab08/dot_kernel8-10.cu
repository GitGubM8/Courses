
#include <hip/hip_runtime.h>
/* ------------
 * This code is provided solely for the personal and private use of 
 * students taking the CSC367 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited. 
 * All forms of distribution of this code, whether as given or with 
 * any changes, are expressly prohibited. 
 * 
 * Authors: Bogdan Simion, Maryam Dehnavi, Felipe de Azevedo Piovezan
 * 
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion and Maryam Dehnavi
 * -------------
*/

__inline__ __device__
float warpReduceSum(float val)
{
    return val;
}

__inline__ __device__
float blockReduceSum(float val)
{
    return val;
}

__global__ void dot_kernel8(float *g_idata1, float *g_idata2, float *g_odata,
        int N)
{
}

__global__ void dot_kernel9(float *g_idata1, float *g_idata2, float *g_odata,
        int N)
{
}

__global__ void dot_kernel10(float *g_idata1, float *g_idata2, float *g_odata,
        int N)
{
}